#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <unistd.h>
#include <stdint.h>
#include <inttypes.h>
#include "microbench_cuda.h"
#define BLOCK_SIZE 1024
extern "C" {
//START


__global__ void baseline_double_cuda_init(double *array, int size) 
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < size) 
    {
        array[pos] = 0;
    }
}

__global__ void baseline_double_cuda_kernel(double *input, double *output, int size, int j) 
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < size)
    {
        output[pos] = input[pos]+pos/(j+1)+input[pos-1];
    }
}


uint64_t *baseline_double_cuda(uint64_t * _array, unsigned long int size)
{
         double *d_array, *d_output;
         hipMalloc((void **) &d_array, sizeof(double)*size);
         hipMalloc((void **) &d_output, sizeof(double)*size);
         hipMemcpy(d_array, _array, sizeof(double)*size, hipMemcpyHostToDevice);

         unsigned int grid_cols = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

         // Launch kernel 
         baseline_double_cuda_init<<<grid_cols, BLOCK_SIZE>>>(d_array, size);
         for(int j=0;j<3;j++)
         {
             baseline_double_cuda_kernel<<<grid_cols, BLOCK_SIZE>>>(d_array, d_output, size, j);
             hipDeviceSynchronize();
             hipMemcpy(d_array, d_output, sizeof(double)*size, hipMemcpyDeviceToDevice);
         }
         // Transefr results from device to host 
         hipMemcpy(_array, d_output, sizeof(double)*size, hipMemcpyDeviceToHost);
         hipFree(d_output);
         hipFree(d_array);
	 return _array;
}
//END

//START_GPU
__global__ void baseline_int_cuda_kernel(uint64_t *input, uint64_t *output, int size, int j) 
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < size)
    {
        output[pos] = input[pos]+pos/(j+1)+input[pos-1];
    }
}



__global__ void baseline_int_cuda_init(uint64_t *array, int size) 
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < size) 
    {
        array[pos] = 0;
    }
}

uint64_t *baseline_int_cuda(uint64_t * _array, unsigned long int size)
{
         uint64_t *d_array, *d_output;
         hipMalloc((void **) &d_array, sizeof(uint64_t)*size);
         hipMalloc((void **) &d_output, sizeof(uint64_t)*size);
         hipMemcpy(d_array, _array, sizeof(uint64_t)*size, hipMemcpyHostToDevice);

         unsigned int grid_cols = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

         // Launch kernel 
         baseline_int_cuda_init<<<grid_cols, BLOCK_SIZE>>>(d_array, size);
         for(int j=0;j<3;j++)
         {
             baseline_int_cuda_kernel<<<grid_cols, BLOCK_SIZE>>>(d_array, d_output, size, j);
             hipDeviceSynchronize();
             hipMemcpy(d_array, d_output, sizeof(uint64_t)*size, hipMemcpyDeviceToDevice);
         }
         // Transefr results from device to host 
         hipMemcpy(_array, d_output, sizeof(uint64_t)*size, hipMemcpyDeviceToHost);
         hipFree(d_output);
         hipFree(d_array);
	 return _array;
}
//END_GPU
}